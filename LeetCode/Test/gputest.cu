﻿
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"
#include <iostream>
//需要将C:\\Program Files (x86)\\Microsoft Visual Studio\\2019\\BuildTools\\VC\\Tools\\MSVC\\14.28.29910\\bin\\Hostx64\\x64
//添加到环境变量中，其实是使用msvc中的cl.exe可执行文件。但是此时的vscode代码提示不仅包含mingw64的提示，还包括msvc的提示，会显示红色波浪线
//但是不影响编译结果和运行(估计是二者选一进行调用)。
//使用nvcc gputest.cu -o gputest.exe 生成可执行文件
int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        std::cout << "使用GPU device " << i << ": " << devProp.name << std::endl;
        std::cout << "设备全局内存总量： " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
        std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
        std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
        std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << devProp.regsPerBlock << std::endl;
        std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
        std::cout << "设备上多处理器的数量： " << devProp.multiProcessorCount << std::endl;
        std::cout << "======================================================" << std::endl;
    }
    return 0;
}
